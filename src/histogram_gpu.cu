#include "hip/hip_runtime.h"
#include <iostream>

#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

#include "common.hpp"
#include "cuda_helper.hpp"

namespace cg = cooperative_groups;
using namespace std;

hipAccessPolicyWindow GetDefalutAccessPolicyWindow()
{
  hipAccessPolicyWindow accessPolicyWindow = {0};
  accessPolicyWindow.base_ptr = (void *)0;
  accessPolicyWindow.num_bytes = 0;
  accessPolicyWindow.hitRatio = 0.f;
  accessPolicyWindow.hitProp = hipAccessPropertyNormal;
  accessPolicyWindow.missProp = hipAccessPropertyStreaming;
  return accessPolicyWindow;
}

void SetHistogramGPUL2Default(const hipStream_t &stream)
{
  CheckCudaErrors(hipDeviceSetLimit(cudaLimitPersistingL2CacheSize, 0));
  hipLaunchAttributeValue stream_attribute;
  stream_attribute.accessPolicyWindow = GetDefalutAccessPolicyWindow();
  // use default stream
  hipStreamSetAttribute(stream, hipLaunchAttributeAccessPolicyWindow, &stream_attribute);
  CheckCudaErrors(cudaCtxResetPersistingL2Cache());
}

void SetHistogramGPUL2PersistingAccesses(const hipStream_t &stream, const uint32_t n_bins, uint32_t *bin_counts)
{
  hipDeviceProp_t prop;
  CheckCudaErrors(hipGetDeviceProperties(&prop, 0));
  cout << "l2CacheSize:" << prop.l2CacheSize << endl;
  cout << "persistingL2CacheMaxSize:" << prop.persistingL2CacheMaxSize << endl;

  const int num_bytes = min(prop.persistingL2CacheMaxSize, (int)sizeof(uint32_t) * n_bins);
  cout << "required persisting L2 cache size:" << num_bytes << endl;
  assert(num_bytes <= prop.persistingL2CacheMaxSize);
  CheckCudaErrors(hipDeviceSetLimit(cudaLimitPersistingL2CacheSize, num_bytes));

  hipLaunchAttributeValue stream_attribute;
  stream_attribute.accessPolicyWindow = GetDefalutAccessPolicyWindow();
  stream_attribute.accessPolicyWindow.base_ptr = (void *)bin_counts;
  const float hit_ratio = 1.f;
  stream_attribute.accessPolicyWindow.num_bytes = num_bytes;
  stream_attribute.accessPolicyWindow.hitRatio = hit_ratio;
  stream_attribute.accessPolicyWindow.hitProp = hipAccessPropertyPersisting;
  stream_attribute.accessPolicyWindow.missProp = hipAccessPropertyStreaming;

  // use default stream
  hipStreamSetAttribute(stream, hipLaunchAttributeAccessPolicyWindow, &stream_attribute);
  CheckCudaErrors(cudaCtxResetPersistingL2Cache());
}

__global__ void HistogramGPUv1Kernel(const int *data, const uint32_t n, uint32_t *bin_counts)
{
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid >= n)
  {
    return;
  }
  const int bin_i = data[tid];
  atomicAdd(bin_counts + bin_i, 1);
}

void HistogramGPUv1(const int *data, const uint32_t n, const uint32_t n_bins, const hipStream_t &stream, uint32_t *bin_counts)
{
  const int threads = 1024;
  const int blocks = (n + threads - 1) / threads;
  CheckCudaErrors(hipMemsetAsync(bin_counts, 0, sizeof(uint32_t) * n_bins, stream));
  HistogramGPUv1Kernel<<<blocks, threads, 0, stream>>>(data, n, bin_counts);
}

__global__ void HistogramGPUv2Kernel(const int *data, const uint32_t n, const uint32_t n_bins, uint32_t *bin_counts)
{
  cg::thread_block cta = cg::this_thread_block();
  extern __shared__ uint32_t s_bin_counts[];
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = gridDim.x * blockDim.x;
  if (threadIdx.x < n_bins)
  {
    s_bin_counts[threadIdx.x] = 0;
  }
  cg::sync(cta);
  for (int i = tid; i < n; i += stride)
  {
    const int bin_i = data[i];
    atomicAdd(s_bin_counts + bin_i, 1);
  }
  cg::sync(cta);
  if (threadIdx.x < n_bins)
  {
    uint32_t sum = s_bin_counts[threadIdx.x];
    atomicAdd(bin_counts + threadIdx.x, sum);
  }
}

void HistogramGPUv2(const int *data, const uint32_t n, const uint32_t n_bins, const hipStream_t &stream, uint32_t *bin_counts)
{
  const int threads = 1024;
  const int blocks = (n + threads - 1) / threads;
  const size_t s_mem_size = sizeof(uint32_t) * n_bins;
  CheckCudaErrors(hipMemsetAsync(bin_counts, 0, sizeof(uint32_t) * n_bins, stream));
  HistogramGPUv2Kernel<<<blocks, threads, s_mem_size, stream>>>(data, n, n_bins, bin_counts);
}

__global__ void HistogramGPUv3Kernel(const int *data, const uint32_t n, const uint32_t n_bins, uint32_t *tmp_bin_counts)
{
  cg::thread_block cta = cg::this_thread_block();
  extern __shared__ uint32_t s_bin_counts[];
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int tmp_bin_counts_offset = blockIdx.x * n_bins;
  const int stride = gridDim.x * blockDim.x;
  if (threadIdx.x < n_bins)
  {
    s_bin_counts[threadIdx.x] = 0;
  }
  cg::sync(cta);
  for (int i = tid; i < n; i += stride)
  {
    const int bin_i = data[i];
    atomicAdd(s_bin_counts + bin_i, 1);
  }
  cg::sync(cta);
  if (threadIdx.x < n_bins)
  {
    uint32_t sum = s_bin_counts[threadIdx.x];
    tmp_bin_counts[tmp_bin_counts_offset + threadIdx.x] = sum;
  }
}

__global__ void HistogramGPUv3MergeKernel(const uint32_t *tmp_bin_counts, const int n, uint32_t *bin_counts)
{
  cg::thread_block cta = cg::this_thread_block();
  extern __shared__ uint32_t s_data[];

  uint32_t sum = 0;
  for (int i = threadIdx.x; i < n; i += blockDim.x)
  {
    sum += tmp_bin_counts[blockIdx.x + i * blockDim.x];
  }
  s_data[threadIdx.x] = sum;
  for (uint stride = blockDim.x / 2; stride > 0; stride >>= 1)
  {
    cg::sync(cta);
    if (threadIdx.x < stride)
    {
      s_data[threadIdx.x] += s_data[threadIdx.x + stride];
    }
  }

  if (threadIdx.x == 0)
  {
    bin_counts[blockIdx.x] = s_data[0];
  }
}

void HistogramGPUv3(const int *data, const uint32_t n, const uint32_t n_bins, const hipStream_t &stream, uint32_t *tmp_bin_counts, uint32_t *bin_counts)
{
  const int threads = 1024;
  const int blocks = min((int)(n + threads - 1) / threads, kMaxBlocks);
  const size_t s_mem_size = sizeof(uint32_t) * n_bins;
  CheckCudaErrors(hipMemsetAsync(bin_counts, 0, sizeof(uint32_t) * n_bins, stream));
  HistogramGPUv3Kernel<<<blocks, threads, s_mem_size, stream>>>(data, n, n_bins, tmp_bin_counts);

  const int merge_threads = 256;
  HistogramGPUv3MergeKernel<<<n_bins, merge_threads, sizeof(uint32_t) * merge_threads, stream>>>(tmp_bin_counts, blocks, bin_counts);
}
